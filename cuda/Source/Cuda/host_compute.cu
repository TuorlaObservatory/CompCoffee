// Copyright 2018 Ilia Kosenkov (Tuorla Observatory, Finland) ilia.kosenkov.at.gm@gmail.com
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy of 
// this software and associated documentation files(the "Software"), to deal 
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or 
// sell copies of the Software, and to permit persons to whom the Software 
// is furnished to do so, subject to the following conditions :
// 
// The above copyright notice and this permission notice shall be included 
// in all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, 
// EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, 
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT
// SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
// WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, 
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.

#include "Includes.h"

using namespace std;

void host_compute(const int n)
{
	std::string message{};

	int* host_a = nullptr;
	int* host_b = nullptr;
	int* host_c = nullptr;
	int* threads = nullptr;
	int* blocks = nullptr;


	int* device_a = nullptr;
	int* device_b = nullptr;
	int* device_c = nullptr;
	
	srand(n);

	try
	{

		if (is_failure(
			hipHostAlloc(&host_a, sizeof(int) * n, hipHostMallocDefault),
			"hipHostAlloc",
			&message))
			throw runtime_error(message);
		if (is_failure(
			hipHostAlloc(&host_b, sizeof(int) * n, hipHostMallocDefault),
			"hipHostAlloc",
			&message))
			throw runtime_error(message);
		if (is_failure(
			hipHostAlloc(&host_c, sizeof(int) * n, hipHostMallocDefault),
			"hipHostAlloc",
			&message))
			throw runtime_error(message);
		if (is_failure(
			hipHostAlloc(&threads, sizeof(int) * n, hipHostMallocDefault),
			"hipHostAlloc",
			&message))
			throw runtime_error(message);
		if (is_failure(
			hipHostAlloc(&blocks, sizeof(int) * n, hipHostMallocDefault),
			"hipHostAlloc",
			&message))
			throw runtime_error(message);

		for(auto i = 0 ; i < n; i++)
		{
			host_a[i] = rand() % 100;
			host_b[i] = rand() % 100;
		}

		if (is_failure(
			hipMalloc(&device_a, sizeof(int) * n),
			"hipMalloc",
			&message))
			throw runtime_error(message);
		if (is_failure(
			hipMalloc(&device_b, sizeof(int) * n),
			"hipMalloc",
			&message))
			throw runtime_error(message);
		if (is_failure(
			hipMalloc(&device_c, sizeof(int) * n),
			"hipMalloc",
			&message))
			throw runtime_error(message);

		if (is_failure(
			hipMemcpy(device_a, host_a, sizeof(int) * n, hipMemcpyHostToDevice),
			"hipMemcpy",
			&message))
			throw runtime_error(message);
		if (is_failure(
			hipMemcpy(device_b, host_b, sizeof(int) * n, hipMemcpyHostToDevice),
			"hipMemcpy",
			&message))
			throw runtime_error(message);

		
		// Run code
		kernel<<< n / 8 + 1, 8>>>(device_a, device_b, device_c, n);

		if (is_failure(
			hipDeviceSynchronize(),
			"hipDeviceSynchronize",
			&message))
			throw runtime_error(message);

		if (is_failure(
			hipMemcpy(threads, device_a, sizeof(int) * n, hipMemcpyDeviceToHost),
			"hipMemcpy",
			&message))
			throw runtime_error(message);
		if (is_failure(
			hipMemcpy(blocks, device_b, sizeof(int) * n, hipMemcpyDeviceToHost),
			"hipMemcpy",
			&message))
			throw runtime_error(message);
		if (is_failure(
			hipMemcpy(host_c, device_c, sizeof(int) * n, hipMemcpyDeviceToHost),
			"hipMemcpy",
			&message))
			throw runtime_error(message);
		
		for (auto i = 0; i < n; i++)
		{
			cout
				<< setw(3) << host_a[i]
				<< " + "
				<< setw(3) << host_b[i]
				<< " = "
				<< setw(4) << host_c[i]
				<< " on thread " << threads[i]
				<< " in group " << blocks[i]
				<< endl;
		}
		cout << endl;

	}
	catch(exception&)
	{
		if (host_a != nullptr)
			hipHostFree(host_a);
		if (host_b != nullptr)
			hipHostFree(host_b);
		if (host_c != nullptr)
			hipHostFree(host_c);
		if (threads != nullptr)
			hipHostFree(threads);
		if (blocks != nullptr)
			hipHostFree(blocks);

		if (device_a != nullptr)
			hipFree(device_a);
		if (device_b != nullptr)
			hipFree(device_b);
		if (device_c != nullptr)
			hipFree(device_c);

		throw;
	}

	if (host_a != nullptr)
		hipHostFree(host_a);
	if (host_b != nullptr)
		hipHostFree(host_b);
	if (host_c != nullptr)
		hipHostFree(host_c);
	if (threads != nullptr)
		hipHostFree(threads);
	if (blocks != nullptr)
		hipHostFree(blocks);

	if (device_a != nullptr)
		hipFree(device_a);
	if (device_b != nullptr)
		hipFree(device_b);
	if (device_c != nullptr)
		hipFree(device_c);
}